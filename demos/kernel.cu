#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/random.h>
#include <iostream>

#define M_PI 3.14159265358979323846
#define M_PI_2 1.57079632679489661923

__global__ void kernel(float4 *framebuffer, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;
    int index = y * width + x;
    framebuffer[index] = make_float4(x / (float)width, y / (float)height, 0.2f, 1.0f);
}

__global__ void kernelSurf(hipSurfaceObject_t surf, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;
    surf2Dwrite(make_float4(x / (float)width, y / (float)height, 0.2f, 1.0f), surf, x * sizeof(float4), y, hipBoundaryModeZero);
}

void renderer(float4 *framebuffer, int width, int height)
{
    dim3 block(32, 32);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    kernel<<<grid, block>>>(framebuffer, width, height);
}

void renderSurf(hipSurfaceObject_t surf, int width, int height)
{
    dim3 block(32, 32);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    kernelSurf<<<grid, block>>>(surf, width, height);
}

#include "../camera.h"
#include "../ray.h"

/**
 * @brief Get the Ray object
 *
 * @param cam the camera, which is used to generate the ray
 * @param uv the uv coordinates, where (0,0) is the bottom left corner and (1,1)
 * @param jitter the jitter, which is used to create a depth of field effect
 * @return ray
 */
__device__ __host__ ray getCamRay(const camera *cam, float2 uv, float2 jitter)
{
    float3 pos = make_float3(cam->pos);
    float3 view = normalize(make_float3(cam->view));
    float3 up = make_float3(cam->up);

    float3 right = normalize(cross(view, up));
    float3 up2 = normalize(cross(right, view));

    float3 dir = normalize(
        view * cam->focalDist +
        right * (uv.x - 0.5f) * tan(cam->fov.x / 360.0 * M_PI) +
        up2 * (uv.y - 0.5f) * tan(cam->fov.y / 360.0 * M_PI));

    float3 jitteredPos =
        pos + right * jitter.x * cam->aperture + up2 * jitter.y * cam->aperture;

    return ray{
        make_float4(jitteredPos, 0.0f),
        make_float4(normalize(jitteredPos + dir * cam->focalDist - pos), 0.0f)};
}

__device__ float intersectSphere(ray mray, const float4 sphere)
{
    float3 o = make_float3(mray.origin);
    float3 d = make_float3(mray.direction);
    float3 center = make_float3(sphere);
    float radius = sphere.w;

    float3 oc = o - center;
    float a = dot(d, d);
    float half_b = dot(oc, d);
    float c = dot(oc, oc) - radius * radius;

    float discriminant = half_b * half_b - a * c;

    if (discriminant < 0)
    {
        return -1.0f;
    }
    else
    {
        return (-half_b - sqrt(discriminant)) / a;
    }
}

__global__ void traceRay(hipSurfaceObject_t surf, int width, int height, const camera *cam, const float4 *spheres, int sphereCount)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x > width || y > height)
        return;

    // generate ray
    float u = x / (float)width;
    float v = y / (float)height;
    float2 uv = make_float2(u, v);
    float2 jitter = make_float2(0.0f, 0.0f);
    ray mray = getCamRay(cam, uv, jitter);

    float depth = 1e20f;
    for (int i = 0; i < sphereCount; i++)
    {
        float t = intersectSphere(mray, spheres[i]);
        if (t > 0.0f && t < depth)
            depth = t;
    }

    float far = 50.0f;
    float near = 0.1f;

    depth = (depth - near) / (far - near);

    surf2Dwrite(make_float4(depth, depth, depth, 1.0f), surf, x * sizeof(float4), y, hipBoundaryModeZero);
}

constexpr int SPHERE_COUNT = 20;

struct SphereGen
{
    thrust::default_random_engine rng;
    thrust::normal_distribution<float> dist;

    SphereGen(float mean, float stddev) : dist(mean, stddev) {}

    __host__ __device__ float4 operator()(const int &i)
    {
        rng.discard(i);
        float3 pos = make_float3(dist(rng), 0, dist(rng));
        float radius = 1.0f + 0.5f * dist(rng);
        return make_float4(pos, radius);
    }
};

void render_frame(hipSurfaceObject_t surf, int width, int height)
{
    // generate spheres
    thrust::device_vector<float4> spheres(SPHERE_COUNT);

    thrust::transform(thrust::device, thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(SPHERE_COUNT), spheres.begin(), SphereGen(0.0f, 5.0f));
    float4 base_sphere = make_float4(0.0f, -1000.0f - 2.0f, 0.0f, 1000.0f);
    spheres[0] = base_sphere;

    camera *cam_d = nullptr;
    hipMalloc(&cam_d, sizeof(camera));
    // generate camera
    float fov = 45.0f;
    camera cam{
        make_float4(0, 5, -10, 0),
        make_float4(0, -0.2, 1, 0),
        make_float4(0, 1, 0, 0),
        make_float2(fov, fov * height / width),
        0.01f,
        0.1f,
    };
    hipMemcpy(cam_d, &cam, sizeof(camera), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    const camera *pCam = cam_d;
    const float4 *pSpheres = thrust::raw_pointer_cast(spheres.data());

    // loop for each pixel
    dim3 block(32, 32);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    traceRay<<<grid, block>>>(surf, width, height, pCam, pSpheres, SPHERE_COUNT);

    hipFree(cam_d);
}

#include "../intersection.h"
#include "bvh.h"

__device__ float intersectTriangle(ray iray, int3 index, const float4 *buffer) {
    float4 v0 = buffer[index.x];
    float4 v1 = buffer[index.y];
    float4 v2 = buffer[index.z];

    // ray-triangle intersection
    float3 e1 = make_float3(v1) - make_float3(v0);
    float3 e2 = make_float3(v2) - make_float3(v0);
    float3 s1 = cross(make_float3(iray.direction), e2);

    float divisor = dot(s1, e1);
    if (divisor == 0.0f)
        return -1.0f;

    float3 d = make_float3(iray.origin) - make_float3(v0);
    float b1 = dot(d, s1) / divisor;
    if (b1 < 0.0f || b1 > 1.0f)
        return -1.0f;

    float3 s2 = cross(d, e1);
    float b2 = dot(make_float3(iray.direction), s2) / divisor;
    if (b2 < 0.0f || b1 + b2 > 1.0f)
        return -1.0f;

    float t = dot(e2, s2) / divisor;
    if (t < 0.0f)
        return -1.0f;

    return 1;
}

__device__ bool ray_aabb_intersect (ray iray, aabb box) {
    float3 invDir = 1.0f / make_float3(iray.direction);
    float3 tbot = invDir * (make_float3(box.upper) - make_float3(iray.origin));
    float3 ttop = invDir * (make_float3(box.lower) - make_float3(iray.origin));

    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), tmin.z);
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), tmax.z);

    return smallest_tmax > largest_tmin;
}

__device__ float ray_aabb_intersect_p (ray iray, aabb box) {
    float3 invDir = 1.0f / make_float3(iray.direction);
    float3 tbot = invDir * (make_float3(box.upper) - make_float3(iray.origin));
    float3 ttop = invDir * (make_float3(box.lower) - make_float3(iray.origin));

    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), tmin.z);
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), tmax.z);

    return smallest_tmax > largest_tmin ? largest_tmin : -1.0f;
}

// closest hit program
__device__ intersection closestHitProgram(const ray &iray, const bvh_tree<int3, float4, true> &tree)
{
    intersection isect;
    isect.distance = -1.0f;

    // traverse the tree
    int stack[64];
    int *stackPtr = stack;
    *stackPtr++ = 0;

    const bvh_node *nodes = tree.nodes;
    const int3 * objects = tree.objects;
    const aabb *aabbs = tree.aabbs;
    const float4 *buffer = tree.buffer;

    do {
        const int idx = *--stackPtr;
        const auto lidx = nodes[idx].left_idx;
        const auto ridx = nodes[idx].right_idx;
        if (ray_aabb_intersect(iray, aabbs[lidx])) {
            const auto obj_idx = nodes[lidx].object_idx;
            if (obj_idx != 0xFFFFFFFF) { // leaf nodes
                const auto obj = objects[obj_idx];
                // const auto t = intersectTriangle(iray, obj, buffer);
                const auto t = ray_aabb_intersect_p(iray, aabbs[lidx]);
                if ( (isect.distance < 0.0f || t < isect.distance)) {
                    isect.distance = t;
                    // isect.index = obj_idx;
                }
            } else { // internal node
                *stackPtr++ = lidx;
            }
        }
        if (ray_aabb_intersect(iray, aabbs[ridx])) {
            const auto obj_idx = nodes[ridx].object_idx;
            if (obj_idx != 0xFFFFFFFF) { // leaf nodes
                const auto obj = objects[obj_idx];
                // const auto t = intersectTriangle(iray, obj, buffer);
                const auto t = ray_aabb_intersect_p(iray, aabbs[ridx]);
                if ( (isect.distance < 0.0f || t < isect.distance)) {
                    isect.distance = t;
                    // isect.index = obj_idx;
                }
            } else { // internal node
                *stackPtr++ = ridx;
            }
        }
    } while (stack != stackPtr);

    return isect;
}
template <typename Index, typename BufferType>
__global__ void traceRay(hipSurfaceObject_t surf, int width, int height, const camera *cam, const bvh_tree<Index, BufferType, true> *tree)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x > width || y > height)
        return;

    // generate ray
    float u = x / (float)width;
    float v = y / (float)height;
    float2 uv = make_float2(u, v);
    float2 jitter = make_float2(0.0f, 0.0f);
    ray mray = getCamRay(cam, uv, jitter);
    
    float depth = 1e20f;

    // ray-triangle intersection
    intersection isect = closestHitProgram(mray, *tree);
    // if (isect.distance > 0.0f) {
        depth = isect.distance;
    // }

    float far = 10.0f;
    float near = 0.1f;

    depth = (depth - near) / (far - near);

    surf2Dwrite(make_float4(depth, depth, depth, 1.0f), surf, x * sizeof(float4), y, hipBoundaryModeZero);
}

void render_frame(hipSurfaceObject_t surf, int width, int height, const bvh_tree<int3, float4, true> &tree) {
    camera *cam_d = nullptr;
    hipMalloc(&cam_d, sizeof(camera));
    // generate camera
    float fov = 45.0f;
    camera cam{
        make_float4(0, 0, -2000, 0),
        make_float4(0, 0, 1, 0),
        make_float4(0, 1, 0, 0),
        make_float2(fov, fov * height / width),
        0.01f,
        0.1f,
    };
    hipMemcpy(cam_d, &cam, sizeof(camera), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    bvh_tree<int3, float4, true> *d_tree;
    hipMalloc(&d_tree, sizeof(bvh_tree<int3, float4, true>));
    hipMemcpy(d_tree, &tree, sizeof(bvh_tree<int3, float4, true>), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    dim3 block(32, 32);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    traceRay<<<grid, block>>>(surf, width, height, cam_d, d_tree);

    hipFree(cam_d);
    hipFree(d_tree);
}